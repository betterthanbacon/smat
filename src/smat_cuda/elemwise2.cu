#include "hip/hip_runtime.h"
#include <smat_cuda/elemwise2.cuh>
#include <smat/vm/instruction_db.h>
#include <smat/vm/util/specialization_table.h>
#include <smat/vm/util/specialization_typelists.h>

SM_NAMESPACE_BEGIN

template <typename T> SM_DEVICE_INLINE T signfd(T x) { return (T)(x > 0) - (T)(x < 0); }
template <typename T> SM_DEVICE_INLINE T signi(T x)  { return (T)(x > 0) - (T)(x < 0); }
template <typename T> SM_DEVICE_INLINE T sqrti(T x)  { return (T)(::sqrtf((typename ctype2ftype(T))x)+0.5f); }
SM_DEVICE_INLINE double clampd(double x)  { return ::max(0.0,::min(1.0,x)); }
SM_DEVICE_INLINE float  lsigf(float  x)   { return 1/(1+::__expf(-x)); }
SM_DEVICE_INLINE double lsigd(double x)   { return 1/(1+::exp(-x)); }

// Define some macros to make, for example, "sqrt(x)" be shorthand for "out(i) = sqrt(arg(i))" 
#define EVAL_AS_X(f) A x = a[i]; x = x; b[j] = (f);
#define EVAL2(types,name,f1,f2)           types(name, EVAL_AS_X(f1), EVAL_AS_X(f2))
#define EVAL3(types,name,f1,f2,f3)        types(name, EVAL_AS_X(f1), EVAL_AS_X(f2), EVAL_AS_X(f3))
#define EVAL4(types,name,f1,f2,f3,f4)     types(name, EVAL_AS_X(f1), EVAL_AS_X(f2), EVAL_AS_X(f3), EVAL_AS_X(f4))
#define EVAL5(types,name,f1,f2,f3,f4,f5)  types(name, EVAL_AS_X(f1), EVAL_AS_X(f2), EVAL_AS_X(f3), EVAL_AS_X(f4), EVAL_AS_X(f5))

EVAL2(DEF_FF,sin,      ::__sinf(x),      ::sin(x))
EVAL2(DEF_FF,cos,      ::__cosf(x),      ::cos(x))
EVAL2(DEF_FF,tan,      ::__tanf(x),      ::tan(x))
EVAL2(DEF_FF,asin,     ::asinf(x),       ::asin(x))
EVAL2(DEF_FF,acos,     ::acosf(x),       ::acos(x))
EVAL2(DEF_FF,atan,     ::atanf(x),       ::atan(x))
EVAL2(DEF_FF,sinh,     ::sinhf(x),       ::sinh(x))
EVAL2(DEF_FF,cosh,     ::coshf(x),       ::cosh(x))
EVAL2(DEF_FF,tanh,     ::tanhf(x),       ::tanh(x))
EVAL2(DEF_FF,asinh,    ::asinhf(x),      ::asinh(x))
EVAL2(DEF_FF,acosh,    ::acoshf(x),      ::acosh(x))
EVAL2(DEF_FF,atanh,    ::atanhf(x),      ::atanh(x))
EVAL2(DEF_FF,exp,      ::__expf(x),      ::exp(x))
EVAL2(DEF_FF,exp2,      ::exp2f(x),     ::exp2(x))
EVAL2(DEF_FF,log,      ::__logf(x),      ::log(x))
EVAL2(DEF_FF,log2,      ::log2f(x),     ::log2(x))
EVAL2(DEF_FF,sigm,   _SM::lsigf(x), _SM::lsigd(x))
EVAL2(DEF_FF,sat ,::__saturatef(x),_SM::clampd(x))
EVAL3(DEF_SS,neg, -x, -x, -x)
EVAL3(DEF_ZZ,not, !x, ~x, ~x)
EVAL5(DEF_GG,abs,  x,    x<0?-x:x,         x,       ::fabsf(x),        ::fabs(x))
EVAL5(DEF_GG,sign, x,    signi(x),      x!=0,        signfd(x),        signfd(x))
EVAL5(DEF_GG,signb,false,   x < 0,         0,       signbit(x),        signbit(x))
EVAL5(DEF_GG,sqrt, x,    sqrti(x),  sqrti(x),       ::sqrtf(x),        ::sqrt(x))
EVAL5(DEF_GG,sqr,  x,         x*x,       x*x,              x*x,              x*x)
EVAL5(DEF_GG,rnd,  x,           x,         x,      ::roundf(x),       ::round(x))
EVAL5(DEF_GG,flr,  x,           x,         x,      ::floorf(x),       ::floor(x))
EVAL5(DEF_GG,ceil, x,           x,         x,       ::ceilf(x),        ::ceil(x))
EVAL5(DEF_GL,isinf,false,   false,     false,(bool)__isinff(x), (bool)__isinf(x))
EVAL5(DEF_GL,isnan,false,   false,     false,(bool)__isnanf(x), (bool)__isnan(x))
EVAL5(DEF_GL,lnot,    !x,    x==0,      x==0,        x == 0.0f,        x == 0.0)

void execute_elemwise2(opcode_t opcode, const argument& a, const argument& b)
{
	#define LAUNCH_CASE(types,matched,f,try_inplace)  \
		if (opcode == oc_##f) { \
			DECL_SPECIALIZATION_TABLE(types,execute_fn2,execute_elemwise2_typed<k_##f,try_inplace>::matched); \
			specialization_table(a.dtype)(opcode,a,b);  \
			return; \
		}
	LAUNCH_CASE(T_F,match,sin,true)
	LAUNCH_CASE(T_F,match,cos,true)
	LAUNCH_CASE(T_F,match,tan,true)
	LAUNCH_CASE(T_F,match,asin,true)
	LAUNCH_CASE(T_F,match,acos,true)
	LAUNCH_CASE(T_F,match,atan,true)
	LAUNCH_CASE(T_F,match,sinh,true)
	LAUNCH_CASE(T_F,match,cosh,true)
	LAUNCH_CASE(T_F,match,tanh,true)
	LAUNCH_CASE(T_F,match,asinh,true)
	LAUNCH_CASE(T_F,match,acosh,true)
	LAUNCH_CASE(T_F,match,atanh,true)
	LAUNCH_CASE(T_F,match,exp,true)
	LAUNCH_CASE(T_F,match,exp2,true)
	LAUNCH_CASE(T_F,match,log,true)
	LAUNCH_CASE(T_F,match,log2,true)
	LAUNCH_CASE(T_F,match,sigm,true)
	LAUNCH_CASE(T_F,match,sat,true)
	LAUNCH_CASE(T_S,match,neg,true)
	LAUNCH_CASE(T_G,match,abs,true)
	LAUNCH_CASE(T_G,match,sign,true)
#if SM_WANT_BOOL || SM_WANT_INT || SM_WANT_UINT 
	LAUNCH_CASE(T_Z,match,not,true)
#endif
	LAUNCH_CASE(T_F,match,signb,true)
	LAUNCH_CASE(T_G,match,sqrt,true)
	LAUNCH_CASE(T_G,match,sqr,true)
	LAUNCH_CASE(T_G,match,rnd,true)
	LAUNCH_CASE(T_G,match,flr,true)
	LAUNCH_CASE(T_G,match,ceil,true)
	LAUNCH_CASE(T_G,logical,isinf,false)
	LAUNCH_CASE(T_G,logical,isnan,false)
	LAUNCH_CASE(T_G,logical,lnot,false)
	SM_UNIMPLEMENTED()
}

SM_NAMESPACE_END
